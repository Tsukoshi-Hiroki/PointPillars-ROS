#include "hip/hip_runtime.h"
// headers in STL
#include <stdio.h>

// headers in local files
#include "common.h"
#include "boolvfe.h"

__global__ void make_pillar_histo_kernel(
    const float* dev_points, 
    float* pillar_count_histo, const int num_points,
    const int grid_x_size, const int grid_y_size, const int grid_z_size, 
    const float min_x_range, const float min_y_range, const float min_z_range, 
    const float pillar_x_size, const float pillar_y_size, const float pillar_z_size,
    const int input_point_feature) {
  int th_i = blockIdx.x * blockDim.x +  threadIdx.x ;
  if (th_i >= num_points) {
    return;
  }
  int x_coor = floor((dev_points[th_i * input_point_feature + 0] - min_x_range) / pillar_x_size);
  int y_coor = floor((dev_points[th_i * input_point_feature + 1] - min_y_range) / pillar_y_size);
  int z_coor = floor((dev_points[th_i * input_point_feature + 2] - min_z_range) / pillar_z_size);

  if (x_coor >= 0 && x_coor < grid_x_size && y_coor >= 0 &&
      y_coor < grid_y_size && z_coor >= 0 && z_coor < grid_z_size) {
    pillar_count_histo[z_coor * grid_x_size * grid_y_size + y_coor * grid_x_size + x_coor] = 1;
  }
}

BoolVFECuda::BoolVFECuda(
    const int num_threads, 
    const int input_point_feature,
    const int num_point_feature,
    const int grid_x_size, const int grid_y_size,
    const int grid_z_size, const float pillar_x_size, const float pillar_y_size,
    const float pillar_z_size, const float min_x_range, const float min_y_range,
    const float min_z_range)
    : num_threads_(num_threads),
      input_point_feature_(input_point_feature),
      num_point_feature_(num_point_feature),
      grid_x_size_(grid_x_size),
      grid_y_size_(grid_y_size),
      grid_z_size_(grid_z_size),
      pillar_x_size_(pillar_x_size),
      pillar_y_size_(pillar_y_size),
      pillar_z_size_(pillar_z_size),
      min_x_range_(min_x_range),
      min_y_range_(min_y_range),
      min_z_range_(min_z_range) {
    
  }

BoolVFECuda::~BoolVFECuda() {
  }

void BoolVFECuda::DoBoolVFECuda(
    const float* dev_points, const int in_num_points, 
    int* dev_x_coors,int* dev_y_coors, 
    float* dev_num_points_per_pillar,
    float* dev_pillar_point_feature, 
    float* dev_pillar_coors,
    int* dev_sparse_pillar_map, 
    int* host_pillar_count, 
    float* dev_pillar_count_histo) {
    // initialize paraments
    std::cout << grid_y_size_ << " " << grid_x_size_ << " " << grid_z_size_ << " " << input_point_feature_ << std::endl;
    GPU_CHECK(hipMemset(dev_pillar_count_histo, 0 , grid_y_size_ * grid_x_size_ * grid_z_size_ * sizeof(float)));
    int num_block = DIVUP(in_num_points , num_threads_);

    make_pillar_histo_kernel<<<num_block , num_threads_>>>(
        dev_points, dev_pillar_count_histo, in_num_points, 
        grid_x_size_, grid_y_size_, grid_z_size_, 
        min_x_range_, min_y_range_, min_z_range_, 
        pillar_x_size_, pillar_y_size_, pillar_z_size_, 
        input_point_feature_);

    // float * temp_float = new float[grid_y_size_ * grid_x_size_ * grid_z_size_](); 
    // GPU_CHECK(hipMemcpy(temp_float, dev_pillar_count_histo, grid_y_size_ * grid_x_size_ * grid_z_size_ * sizeof(float), hipMemcpyDeviceToHost));
    // int cnt = 0;
    // for (int i = 0; i < grid_y_size_ * grid_x_size_ * grid_z_size_; i++){
    //     cnt += temp_float[i];
    // } 
    // std::cout << cnt << std::endl;
}